#include <stdio.h>
#include <hip/hip_runtime.h>

struct CudaMatrix
{
    float *data;
};

extern "C"
{
    int init_cuda_matrix(CudaMatrix *cuda_matrix, const float *data, int data_size)
    {
        hipMalloc((void **)&cuda_matrix->data, data_size * sizeof(float));
        hipMemcpy(cuda_matrix->data, data, data_size * sizeof(float), hipMemcpyHostToDevice);
        return 0;
    }

    int free_cuda_matrix(CudaMatrix *cuda_matrix)
    {
        hipFree(cuda_matrix->data);
        return 0;
    }
}
